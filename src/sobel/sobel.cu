#include "hip/hip_runtime.h"
#include "sobel.h"
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

namespace chrono = std::chrono;

__constant__ int d_kernel_x[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int d_kernel_y[3][3] = {
    {-1, -2, -1},
    {0, 0, 0},
    {1, 2, 1}
};

__global__ void sobelKernel(float* input, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1) {
        return;
    }

    float sum_x = 0;
    float sum_y = 0;

    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            sum_x += d_kernel_x[i + 1][j + 1] * input[(y + i) * width + (x + j)];
            sum_y += d_kernel_y[i + 1][j + 1] * input[(y + i) * width + (x + j)];
        }
    }

    float magnitude = sqrtf(sum_x * sum_x + sum_y * sum_y);
    output[(y-1) * width + (x-1)] = fminf(255.0f, magnitude);
}

void sobelCUDA(GrayImage* image) {
    int width = image->width;
    int height = image->height;
    int size = width * height * sizeof(float);
    int new_size = (width-2) * (height-2) * sizeof(float);

    float* d_input;
    float* d_output;
    float* input = new float[size];
    float* result = new float[new_size];

    for(int i = 0; i < height; i++) {
	memcpy(input+i*width, image->image[i], width*sizeof(float));
    }

    // Error checking for hipMalloc
    if (hipMalloc(&d_input, size) != hipSuccess) {
        std::cerr << "Failed to allocate device memory for input." << std::endl;
        return;
    }
    if (hipMalloc(&d_output, new_size) != hipSuccess) {
        std::cerr << "Failed to allocate device memory for output." << std::endl;
        hipFree(d_input);
        return;
    }

    // Error checking for hipMemcpy
    if (hipMemcpy(d_input, input, size, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Failed to copy data to device memory." << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    sobelKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    // Error checking for kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Error checking for hipMemcpy
    if (hipMemcpy(result, d_output, new_size, hipMemcpyDeviceToHost) != hipSuccess) {
        std::cerr << "Failed to copy data from device memory." << std::endl;
    }
    int new_height = height - 2;
    int new_width = width - 2;
    for(int i = 0; i < new_height; i++) {
	memcpy(image->image[i], result+i*new_width, new_width*sizeof(float));
    }

    hipFree(d_input);
    hipFree(d_output);

    image->width = width - 2;
    image->height = height - 2;
}

int main(int argc, char** argv) {
    bool verbose = false;
    if (argc > 1) {
        auto arg1 = std::string(argv[1]);
        if (arg1 == "-v" || arg1 == "--verbose") {
            verbose = true;
        }
    }

    std::cout << "========== CUDA Sobel ==========" << std::endl;
    std::cout << "Loading images..." << std::endl;

    std::vector<GrayImage*> images = getBSDS500Images(verbose);

    std::cout << "Start processing images..." << std::endl;

    auto start = chrono::high_resolution_clock::now();
    for (auto& image : images) {
        if (verbose) {
            std::cout << "Processing image ["
                << image->file_name << "]..." << std::endl;
        }
        sobelCUDA(image);

        image->saveImage("../sobel_outputs/cuda");
        if (verbose) {
            std::cout << "Saved output of image [" 
                << image->file_name << "] successfully" << std::endl;
        }
        delete image;
    }
    auto end = chrono::high_resolution_clock::now();

    auto duration = chrono::duration_cast<chrono::nanoseconds>(end - start);
    std::cout << "Duration: " << duration.count() << " ns" << std::endl;

    return 0;
}

