#include "hip/hip_runtime.h"
#include "sobel.h"
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

namespace chrono = std::chrono;

__constant__ int s_kernel_x[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

__constant__ int s_kernel_y[3][3] = {
    {-1, -2, -1},
    {0, 0, 0},
    {1, 2, 1}
};

__global__ void sobelKernel(float* input, float* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1) {
        return;
    }

    float sum_x = 0;
    float sum_y = 0;

    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            sum_x += s_kernel_x[i + 1][j + 1] * input[(y + i) * width + (x + j)];
            sum_y += s_kernel_y[i + 1][j + 1] * input[(y + i) * width + (x + j)];
        }
    }

    float magnitude = sqrtf(sum_x * sum_x + sum_y * sum_y);
    output[y * width + x] = fminf(255.0f, magnitude);
}

void sobelCUDA(GrayImage* image) {
    int width = image->width;
    int height = image->height;
    int size = width * height * sizeof(float);

    float* d_input;
    float* d_output;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, image->image, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    sobelKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    hipMemcpy(image->image, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char** argv) {
    bool verbose = false;
    if (argc > 1) {
        auto arg1 = std::string(argv[1]);
        if (arg1 == "-v" || arg1 == "--verbose") {
            verbose = true;
        }
    }

    std::cout << "========== CUDA Sobel ==========" << std::endl;
    std::cout << "Loading images..." << std::endl;

    std::string image_path = "../inputs_BSDS500/BSDS500/data/images/";
    auto test = getInputImages(image_path + "test", verbose);
    auto train = getInputImages(image_path + "train", verbose);
    auto val = getInputImages(image_path + "val", verbose);

    std::vector<GrayImage*> images;
    images.insert(images.end(), test.begin(), test.end());
    images.insert(images.end(), train.begin(), train.end());
    images.insert(images.end(), val.begin(), val.end());

    std::cout << "Start processing images..." << std::endl;

    auto start = chrono::high_resolution_clock::now();
    for (auto& image : images) {
        if (verbose) {
            std::cout << "Processing image ["
                << image->file_name << "]..." << std::endl;
        }
        sobelCUDA(image);

        image->saveImage("../sobel_outputs/cuda");
        if (verbose) {
            std::cout << "Saved output of image [" 
                << image->file_name << "] successfully" << std::endl;
        }
        delete image;
    }
    auto end = chrono::high_resolution_clock::now();

    auto duration = chrono::duration_cast<chrono::nanoseconds>(end - start);
    std::cout << "Duration: " << duration.count() << " ns" << std::endl;

    return 0;
}
