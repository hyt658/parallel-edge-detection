#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "canny.h"

__global__ void gaussianFilterKernel(
    float* d_image, float* d_new_image, int width, int height, float* d_kernel
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kernel_radius = gaussian_kernel_size / 2;
    int x_bound = width - kernel_radius;
    int y_bound = height - kernel_radius;

    if (x < kernel_radius || x >= x_bound || y < kernel_radius || y >= y_bound) {
        return;
    }

    float magnitude = 0.0f;
    for (int i = -kernel_radius; i <= kernel_radius; i++) {
        for (int j = -kernel_radius; j <= kernel_radius; j++) {
            int img_idx = (y + i) * width + (x + j);
            int kernel_idx =
                (i + kernel_radius) * gaussian_kernel_size + (j + kernel_radius);
            magnitude += d_image[img_idx] * d_kernel[kernel_idx];
        }
    }

    int new_image_idx =
        (y - kernel_radius) * (width - kernel_radius*2) + (x - kernel_radius);
    d_new_image[new_image_idx] = magnitude;
}

__global__ void computeGradientKernel(
    float* d_image, float* d_new_image, float* d_direction, int width, int height,
    int* d_sobel_x, int* d_sobel_y
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kernel_radius = sobel_kernel_size / 2;
    int x_bound = width - kernel_radius;
    int y_bound = height - kernel_radius;

    if (x < kernel_radius || x >= x_bound || y < kernel_radius || y >= y_bound) {
        return;
    }

    float sum_x = 0.0f;
    float sum_y = 0.0f;
    for (int i = -kernel_radius; i <= kernel_radius; ++i) {
        for (int j = -kernel_radius; j <= kernel_radius; ++j) {
            int img_idx = (y + i) * width + (x + j);
            int kernel_idx =
                (i + kernel_radius) * sobel_kernel_size + (j + kernel_radius);
            sum_x += d_image[img_idx] * d_sobel_x[kernel_idx];
            sum_y += d_image[img_idx] * d_sobel_y[kernel_idx];
        }
    }

    int new_image_idx =
        (y - kernel_radius) * (width - kernel_radius*2) + (x - kernel_radius);
    d_new_image[new_image_idx] = sqrtf(sum_x * sum_x + sum_y * sum_y);
    d_direction[new_image_idx] = atan2f(sum_y, sum_x) * 180.0f / HIP_PI;
}

__global__ void nonMaxSuppression(
    float* d_image, float* d_direction, float* d_new_image, int width, int height
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= 0 || x >= width-1 || y <= 0 || y >= height-1) {
        return;
    }

    float direction = d_direction[y*width+x];
    float magnitude = d_image[y*width+x];
    float first_pixel = 0.0f;
    float second_pixel = 0.0f;

    if ((direction >= -22.5f && direction < 22.5f) || 
        (direction >= 157.5f / 8 && direction < -157.5f)) {
        // fall in 0 degree direction area
        first_pixel = d_image[y*width+x-1];
        second_pixel = d_image[y*width+x+1];
    } else if ((direction >= 22.5f && direction < 67.5f) ||
               (direction >= -157.5f && direction < -112.5f)) {
        // fall in 45 degree direction area
        first_pixel = d_image[(y-1)*width+x-1];
        second_pixel = d_image[(y+1)*width+x+1];
    } else if ((direction >= 67.5f && direction < 112.5f) ||
               (direction >= -112.5f && direction < -67.5f)) {
        // fall in 90 degree direction area
        first_pixel = d_image[(y-1)*width+x];
        second_pixel = d_image[(y+1)*width+x];
    } else if ((direction >= 112.5f && direction < 157.5f) ||
               (direction >= -67.5f && direction < -22.5f)) {
        // fall in 135 degree direction area
        first_pixel = d_image[(y-1)*width+x+1];
        second_pixel = d_image[(y+1)*width+x-1];
    }

    if (magnitude >= first_pixel && magnitude >= second_pixel) {
        d_new_image[y*width+x] = magnitude;
    } else {
        d_new_image[y*width+x] = 0.0f;
    }
}

__global__ void doubleThresholdKernel(
    float* d_image, float* d_new_image, int width, int height,
    float low_threshold, float high_threshold
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int idx = y * width + x;
    float magnitude = d_image[idx];

    if (magnitude >= high_threshold) {
        d_new_image[idx] = 255.0f;
    } else if (magnitude >= low_threshold) {
        bool found_strong = false;
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                if (y + dy < 0 || y + dy >= height ||
                    x + dx < 0 || x + dx >= width) {
                    continue;
                }
                int img_idx = (y + dy) * width + x + dx;
                if (d_image[img_idx] >= high_threshold) {
                    found_strong = true;
                    break;
                }
            }
            if (found_strong) { break; }
        }

        if (found_strong) {
            d_new_image[idx] = 255.0f;
        } else {
            d_new_image[idx] = 0.0f;
        }
    } else {
        d_new_image[idx] = 0.0f;
    }
}

void cannyCUDA(GrayImage* image) {
    int width = image->width;
    int height = image->height;
    int size = width * height;
    float* linear_image = new float[size];
    for (int i = 0; i < height; ++i) {
        float* dest_pos = linear_image + i * width;
        memcpy(dest_pos, image->image[i], width * sizeof(float));
        delete[] image->image[i];
    }
    delete[] image->image;

    // generate gaussian kernel
    float sum = 0.0f;
    int linear_gaussian_size = gaussian_kernel_size * gaussian_kernel_size;
    int gaussian_kernel_radius = gaussian_kernel_size / 2;
    float* gaussian_kernel = new float[linear_gaussian_size];
    for (int y = -gaussian_kernel_radius; y <= gaussian_kernel_radius; ++y) {
        int y_idx = y + gaussian_kernel_radius;
        for (int x = -gaussian_kernel_radius; x <= gaussian_kernel_radius; ++x) {
            int x_idx = x + gaussian_kernel_radius;
            float temp = exp(-(x * x + y * y) / (2 * gaussian_sd * gaussian_sd)) / 
                (2 * M_PI * gaussian_sd * gaussian_sd);
            sum += temp;
            gaussian_kernel[y_idx * gaussian_kernel_size + x_idx] = temp;
        }
    }
    // normalize gaussian kernel
    for (int i = 0; i < linear_gaussian_size; ++i) {
        gaussian_kernel[i] /= sum;
    }

    int linear_sobel_size = sobel_kernel_size * sobel_kernel_size;
    int* linear_sobel_x = new int[linear_sobel_size];
    int* linear_sobel_y = new int[linear_sobel_size];
    for (int y = 0; y < sobel_kernel_size; ++y) {
        int* dest_pos_x = linear_sobel_x + y * sobel_kernel_size;
        int* dest_pos_y = linear_sobel_y + y * sobel_kernel_size;
        memcpy(dest_pos_x, sobel_x[y], sobel_kernel_size * sizeof(int));
        memcpy(dest_pos_y, sobel_y[y], sobel_kernel_size * sizeof(int));
    }

    float* d_image = nullptr;
    float* d_new_image = nullptr;
    float* d_direction = nullptr;
    int* d_sobel_x = nullptr;
    int* d_sobel_y = nullptr;
    float* d_gaussian_kernel = nullptr;

    hipMalloc(&d_image, size*sizeof(float));
    hipMalloc(&d_new_image, size*sizeof(float));
    hipMalloc(&d_direction, size*sizeof(float));
    hipMalloc(&d_sobel_x, linear_sobel_size*sizeof(int));
    hipMalloc(&d_sobel_y, linear_sobel_size*sizeof(int));
    hipMalloc(&d_gaussian_kernel, linear_gaussian_size*sizeof(float));
    hipMemcpy(d_image, linear_image, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sobel_x, linear_sobel_x, 
        linear_sobel_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sobel_y, linear_sobel_y,
        linear_sobel_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_gaussian_kernel, gaussian_kernel, 
        linear_gaussian_size*sizeof(float), hipMemcpyHostToDevice);
    

    int block_x = 16;
    int block_y = 16;
    int grid_x = (width + block_x - 1) / block_x;
    int grid_y = (height + block_y - 1) / block_y;
    dim3 block(block_x, block_y);
    dim3 grid(grid_x, grid_y);

    gaussianFilterKernel<<<grid, block>>>
        (d_image, d_new_image, width, height, d_gaussian_kernel);
    hipDeviceSynchronize();
    width = getOutputWidth(width, gaussian_kernel_size);
    height = getOutputHeight(height, gaussian_kernel_size);
    size = width * height;
    hipMemcpy(d_image, d_new_image, size*sizeof(float), hipMemcpyDeviceToDevice);

    computeGradientKernel<<<grid, block>>>
        (d_image, d_new_image, d_direction, width, height, d_sobel_x, d_sobel_y);
    hipDeviceSynchronize();
    width = getOutputWidth(width, sobel_kernel_size);
    height = getOutputHeight(height, sobel_kernel_size);
    size = width * height;
    hipMemcpy(d_image, d_new_image, size*sizeof(float), hipMemcpyDeviceToDevice);

    nonMaxSuppression<<<grid, block>>>
        (d_image, d_direction, d_new_image, width, height);
    hipDeviceSynchronize();
    hipMemcpy(d_image, d_new_image, size*sizeof(float), hipMemcpyDeviceToDevice);

    doubleThresholdKernel<<<grid, block>>>
        (d_image, d_new_image, width, height, low_threshold, high_threshold);
    hipDeviceSynchronize();
    hipMemcpy(linear_image, d_new_image, size*sizeof(float), hipMemcpyDeviceToHost);

    float** new_image_2d = new float*[height];
    for (int y = 0; y < height; ++y) {
        new_image_2d[y] = new float[width];
        float* src_pos = linear_image + y * width;
        memcpy(new_image_2d[y], src_pos, width * sizeof(float));
    }

    image->image = new_image_2d;
    image->width = width;
    image->height = height;

    delete[] linear_image;
    delete[] gaussian_kernel;
    hipFree(d_image);
    hipFree(d_new_image);
    hipFree(d_direction);
    hipFree(d_sobel_x);
    hipFree(d_sobel_y);
    hipFree(d_gaussian_kernel);
}

int main(int argc, char** argv) {
    bool verbose = false;
    if (argc > 1) {
        auto arg1 = std::string(argv[1]);
        if (arg1 == "-v" || arg1 == "--verbose") {
            verbose = true;
        }
    }

    std::cout << "==========CUDA Canny==========" << std::endl;
    std::cout << "Loading images..." << std::endl;
    std::vector<GrayImage*> images = getBSDS500Images(verbose);

    std::cout << "Start processing images..." << std::endl;
    auto start = chrono::high_resolution_clock::now();
    for (auto& image : images) {
        if (verbose) {
            std::cout << "Processing image ["
                << image->file_name << "]..." << std::endl;
        }
        cannyCUDA(image);

        image->saveImage("../canny_outputs/cuda");
        if (verbose) {
            std::cout << "Saved output of image [" 
                << image->file_name << "] successfully" << std::endl;
        }
        delete image;
    }
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::nanoseconds>(end - start);
    std::cout << "Duration: " << duration.count() << " ns" << std::endl;

    return 0;
}
